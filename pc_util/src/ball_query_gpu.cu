#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>
#include <stdlib.h>


#include "ball_query_gpu.h"
#include "cuda_utils.h"


__global__ void ball_query_kernel_fast(int b, int n, int m, float radius, int nsample,
    const float *__restrict__ new_xyz, const float *__restrict__ xyz, int *__restrict__ idx) {
    // new_xyz: (B, M, 3)
    // xyz: (B, N, 3)
    // output:
    //      idx: (B, M, nsample)
    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= m) return;

    new_xyz += bs_idx * m * 3 + pt_idx * 3;
    xyz += bs_idx * n * 3;
    idx += bs_idx * m * nsample + pt_idx * nsample;

    float radius2 = radius * radius;
    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];

    int cnt = 0;
    for (int k = 0; k < n; ++k) {
        float x = xyz[k * 3 + 0];
        float y = xyz[k * 3 + 1];
        float z = xyz[k * 3 + 2];
        float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) + (new_z - z) * (new_z - z);
        if (d2 < radius2){
            if (cnt == 0){
                for (int l = 0; l < nsample; ++l) {
                    idx[l] = k;
                }
            }
            idx[cnt] = k;
            ++cnt;
            if (cnt >= nsample) break;
        }
    }
}


void ball_query_kernel_launcher_fast(int b, int n, int m, float radius, int nsample, \
    const float *new_xyz, const float *xyz, int *idx) {
    // new_xyz: (B, M, 3)
    // xyz: (B, N, 3)
    // output:
    //      idx: (B, M, nsample)

    hipError_t err;

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    ball_query_kernel_fast<<<blocks, threads>>>(b, n, m, radius, nsample, new_xyz, xyz, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void ball_center_query_kernel_fast(int b, int n, int m, float radius, \
    const float *__restrict__ point, const float *__restrict__ key_point, int *__restrict__ idx) {
    // key_point: (B, M, 3)
    // point: (B, N, 3)
    // output:
    //      idx: (B, N)
    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= n) return;

    point += bs_idx * n * 3 + pt_idx * 3;
    key_point += bs_idx * m * 3;
    idx += bs_idx * n + pt_idx;

    float radius2 = radius * radius;
    float point_x = point[0];
    float point_y = point[1];
    float point_z = point[2];

    float bestd = 1e8;
    for (int k = 0; k < m; ++k) {
        float x = key_point[k * 3 + 0];
        float y = key_point[k * 3 + 1];
        float z = key_point[k * 3 + 2];
        if (((x + 1) * (x + 1) + (y + 1) * (y + 1) + (z + 1) * (z + 1)) < 1e-4) break;
        float d2 = (point_x - x) * (point_x - x) + (point_y - y) * (point_y - y) + (point_z - z) * (point_z - z);
        if (d2 < radius2 && d2 < bestd){
            idx[0] = k;
            bestd = d2;
        }
    }
}


void ball_center_query_kernel_launcher_fast(int b, int n, int m, float radius, \
    const float *point, const float *key_point, int *idx) {
    // point: (B, n, 3)
    // key_point: (B, m, 3)
    // output:
    //      idx: (B, n)

    hipError_t err;

    dim3 blocks(DIVUP(n, THREADS_PER_BLOCK), b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    ball_center_query_kernel_fast<<<blocks, threads>>>(b, n, m, radius, point, key_point, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}





__global__ void knn_query_kernel_fast(int b, int n, int m, int nsample, const float *__restrict__ new_xyz,
    const float *__restrict__ xyz, float *__restrict__ dist2, int *__restrict__ idx) {

    // new_xyz: (B, M, 3)
    // xyz: (B, N, 3)
    // output:
    //      dist2: (B, M, nsample)
    //      idx: (B, M, nsample)

    int bs_idx = blockIdx.y;
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (bs_idx >= b || pt_idx >= m) return;

    new_xyz += bs_idx * m * 3 + pt_idx * 3;
    xyz += bs_idx * n * 3;
    dist2 += bs_idx * m * nsample + pt_idx * nsample;
    idx += bs_idx * m * nsample + pt_idx * nsample;

    float nx = new_xyz[0];
    float ny = new_xyz[1];
    float nz = new_xyz[2];

    for (int i = 0; i < n; ++i) {
        float x = xyz[i * 3 + 0];
        float y = xyz[i * 3 + 1];
        float z = xyz[i * 3 + 2];
        float d2 = (nx - x) * (nx - x) + (ny - y) * (ny - y) + (nz - z) * (nz - z);
        if (d2 < dist2[nsample - 1]) {
            dist2[nsample - 1] = d2;
            idx[nsample - 1] = i;
            for (int j = nsample - 2; j >= 0; j--) {
                if (d2 < dist2[j]){
                    dist2[j + 1] = dist2[j];
                    dist2[j] = d2;
                    idx[j + 1] = idx[j];
                    idx[j] = i;
                }
            }
        }
    }
}


void knn_query_kernel_launcher_fast(int b, int n, int m, int nsample, \
    const float *new_xyz, const float *xyz, float *dist2, int *idx) {
    hipError_t err;

    dim3 blocks(DIVUP(m, THREADS_PER_BLOCK), b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    knn_query_kernel_fast<<<blocks, threads>>>(b, n, m, nsample, new_xyz, xyz, dist2, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}








__global__ void ball_query_kernel_stack(int B, int M, float radius, int nsample, \
    const float *new_xyz, const int *new_xyz_batch_cnt, const float *xyz, const int *xyz_batch_cnt, int *idx) {
    // :param xyz: (N1 + N2 ..., 3) xyz coordinates of the features
    // :param xyz_batch_cnt: (batch_size), [N1, N2, ...]
    // :param new_xyz: (M1 + M2 ..., 3) centers of the ball query
    // :param new_xyz_batch_cnt: (batch_size), [M1, M2, ...]
    // output:
    //      idx: (M, nsample)
    int pt_idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (pt_idx >= M) return;

    int bs_idx = 0, pt_cnt = new_xyz_batch_cnt[0];
    for (int k = 1; k < B; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += new_xyz_batch_cnt[k];
        bs_idx = k;
    }

    int xyz_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) xyz_batch_start_idx += xyz_batch_cnt[k];
    // for (int k = 0; k < bs_idx; k++) new_xyz_batch_start_idx += new_xyz_batch_cnt[k];

    new_xyz += pt_idx * 3;
    xyz += xyz_batch_start_idx * 3;
    idx += pt_idx * nsample;

    float radius2 = radius * radius;
    float new_x = new_xyz[0];
    float new_y = new_xyz[1];
    float new_z = new_xyz[2];
    int n = xyz_batch_cnt[bs_idx];

    int cnt = 0;
    for (int k = 0; k < n; ++k) {
        float x = xyz[k * 3 + 0];
        float y = xyz[k * 3 + 1];
        float z = xyz[k * 3 + 2];
        float d2 = (new_x - x) * (new_x - x) + (new_y - y) * (new_y - y) + (new_z - z) * (new_z - z);
        if (d2 < radius2){
            if (cnt == 0){
                for (int l = 0; l < nsample; ++l) {
                    idx[l] = k;
                }
            }
            idx[cnt] = k;
            ++cnt;
            if (cnt >= nsample) break;
        }
    }
    if (cnt == 0) idx[0] = -1;
}


void ball_query_kernel_launcher_stack(int B, int M, float radius, int nsample,
    const float *new_xyz, const int *new_xyz_batch_cnt, const float *xyz, const int *xyz_batch_cnt, int *idx){
    // :param xyz: (N1 + N2 ..., 3) xyz coordinates of the features
    // :param xyz_batch_cnt: (batch_size), [N1, N2, ...]
    // :param new_xyz: (M1 + M2 ..., 3) centers of the ball query
    // :param new_xyz_batch_cnt: (batch_size), [M1, M2, ...]
    // output:
    //      idx: (M, nsample)

    hipError_t err;

    dim3 blocks(DIVUP(M, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    ball_query_kernel_stack<<<blocks, threads>>>(B, M, radius, nsample, new_xyz, new_xyz_batch_cnt, xyz, xyz_batch_cnt, idx);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
