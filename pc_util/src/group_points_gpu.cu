#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "cuda_utils.h"
#include "group_points_gpu.h"


__global__ void group_points_grad_kernel_fast(int b, int c, int n, int npoints, int nsample, 
    const float *__restrict__ grad_out, const int *__restrict__ idx, float *__restrict__ grad_points) {
    // grad_out: (B, C, npoints, nsample)
    // idx: (B, npoints, nsample)
    // output:
    //      grad_points: (B, C, N)
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_idx = index / nsample;
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int sample_idx = index % nsample;
    grad_out += bs_idx * c * npoints * nsample + c_idx * npoints * nsample + pt_idx * nsample + sample_idx;
    idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx; 
    
    atomicAdd(grad_points + bs_idx * c * n + c_idx * n + idx[0] , grad_out[0]);
}

void group_points_grad_kernel_launcher_fast(int b, int c, int n, int npoints, int nsample, 
    const float *grad_out, const int *idx, float *grad_points) {
    // grad_out: (B, C, npoints, nsample)
    // idx: (B, npoints, nsample)
    // output:
    //      grad_points: (B, C, N)
    hipError_t err;
    dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_grad_kernel_fast<<<blocks, threads>>>(b, c, n, npoints, nsample, grad_out, idx, grad_points);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void group_points_kernel_fast(int b, int c, int n, int npoints, int nsample, 
    const float *__restrict__ points, const int *__restrict__ idx, float *__restrict__ out) {
    // points: (B, C, N)
    // idx: (B, npoints, nsample)
    // output:
    //      out: (B, C, npoints, nsample)
    int bs_idx = blockIdx.z;
    int c_idx = blockIdx.y;
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int pt_idx = index / nsample;
    if (bs_idx >= b || c_idx >= c || pt_idx >= npoints) return;

    int sample_idx = index % nsample;

    idx += bs_idx * npoints * nsample + pt_idx * nsample + sample_idx; 
    int in_idx = bs_idx * c * n + c_idx * n + idx[0];
    int out_idx = bs_idx * c * npoints * nsample + c_idx * npoints * nsample + pt_idx * nsample + sample_idx;

    out[out_idx] = points[in_idx];
}


void group_points_kernel_launcher_fast(int b, int c, int n, int npoints, int nsample, 
    const float *points, const int *idx, float *out) {
    // points: (B, C, N)
    // idx: (B, npoints, nsample)
    // output:
    //      out: (B, C, npoints, nsample)
    hipError_t err;
    dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_kernel_fast<<<blocks, threads>>>(b, c, n, npoints, nsample, points, idx, out);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void group_points_grad_kernel_stack(int B, int M, int C, int N, int nsample,
    const float *grad_out, const int *idx, const int *idx_batch_cnt, const int *features_batch_cnt, float *grad_features) {
    // :param grad_out: (M1 + M2 ..., C, nsample) tensor of the gradients of the output from forward
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :return:
    //     grad_features: (N1 + N2 ..., C) gradient of the features
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sample_idx = index % nsample;
    int C_idx = (index / nsample) % C;
    int pt_idx = (index / nsample / C);

    if (pt_idx >= M || C_idx >= C || sample_idx >= nsample) return;

    int bs_idx = 0, pt_cnt = idx_batch_cnt[0];
    for (int k = 1; k < B; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += idx_batch_cnt[k];
        bs_idx = k;
    }

    int features_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) features_batch_start_idx += features_batch_cnt[k];

    grad_out += pt_idx * C * nsample + C_idx * nsample + sample_idx;
    idx += pt_idx * nsample + sample_idx;
    grad_features += (features_batch_start_idx + idx[0]) * C + C_idx;

    atomicAdd(grad_features, grad_out[0]);
}

void group_points_grad_kernel_launcher_stack(int B, int M, int C, int N, int nsample,
    const float *grad_out, const int *idx, const int *idx_batch_cnt, const int *features_batch_cnt, float *grad_features) {
    // :param grad_out: (M1 + M2 ..., C, nsample) tensor of the gradients of the output from forward
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :return:
    //     grad_features: (N1 + N2 ..., C) gradient of the features

    hipError_t err;
    // dim3 blocks(DIVUP(npoints * nsample, THREADS_PER_BLOCK), c, b);  // blockIdx.x(col), blockIdx.y(row)
    dim3 blocks(DIVUP(M * C * nsample, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_grad_kernel_stack<<<blocks, threads>>>(B, M, C, N, nsample, grad_out, idx, idx_batch_cnt, features_batch_cnt, grad_features);

    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}


__global__ void group_points_kernel_stack(int B, int M, int C, int nsample,
    const float *features, const int *features_batch_cnt, const int *idx, const int *idx_batch_cnt, float *out) {
    // :param features: (N1 + N2 ..., C) tensor of features to group
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :return:
    //     output: (M1 + M2, C, nsample) tensor
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int sample_idx = index % nsample;
    int C_idx = (index / nsample) % C;
    int pt_idx = (index / nsample / C);

    if (pt_idx >= M || C_idx >= C || sample_idx >= nsample) return;

    int bs_idx = 0, pt_cnt = idx_batch_cnt[0];
    for (int k = 1; k < B; k++){
        if (pt_idx < pt_cnt) break;
        pt_cnt += idx_batch_cnt[k];
        bs_idx = k;
    }

    int features_batch_start_idx = 0;
    for (int k = 0; k < bs_idx; k++) features_batch_start_idx += features_batch_cnt[k];
    features += features_batch_start_idx * C;

    idx += pt_idx * nsample + sample_idx;
    int in_idx = idx[0] * C + C_idx;
    int out_idx = pt_idx * C * nsample + C_idx * nsample + sample_idx;

    out[out_idx] = features[in_idx];
}


void group_points_kernel_launcher_stack(int B, int M, int C, int nsample,
    const float *features, const int *features_batch_cnt, const int *idx, const int *idx_batch_cnt, float *out) {
    // :param features: (N1 + N2 ..., C) tensor of features to group
    // :param features_batch_cnt: (batch_size) [N1 + N2 ...] tensor containing the indicies of features to group with
    // :param idx: (M1 + M2 ..., nsample) tensor containing the indicies of features to group with
    // :param idx_batch_cnt: (batch_size) [M1 + M2 ...] tensor containing the indicies of features to group with
    // :return:
    //     output: (M1 + M2, C, nsample) tensor

    hipError_t err;
    dim3 blocks(DIVUP(M * C * nsample, THREADS_PER_BLOCK));  // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    group_points_kernel_stack<<<blocks, threads>>>(B, M, C, nsample, features, features_batch_cnt, idx, idx_batch_cnt, out);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
        fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
        exit(-1);
    }
}
